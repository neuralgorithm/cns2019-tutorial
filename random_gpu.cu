#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <SFMT.h>

#define N 4000
#define N_EXC 3200
#define N_INH ( ( N ) - ( N_EXC ) )

#define T 1000.
#define DT 1.

#define TAU_M 20.
#define TAU_GE 5.
#define TAU_GI 10.

#define V_LEAK -49.
#define V_INIT -60.
#define V_RESET -60.
#define THETA -50.

#define G_EXC 1.62
#define G_INH -9.
#define P 0.02

#define BLOCK_SIZE 32

extern "C" { void sfmt_init_gen_rand ( sfmt_t * sfmt, uint32_t seed ); }
extern "C" { double sfmt_genrand_real2 ( sfmt_t * sfmt ); }
extern "C" { void timer_start ( void ); }
extern "C" { double timer_elapsed ( void ); }

static double v [ N ], ge [ N ], gi [ N ];
static int *w_exc, *w_inh, spike [ N ];
static double *d_v, *d_ge, *d_gi;
static int *d_w_exc, *d_w_inh, *d_spike;;

static FILE *file_spike;

void initialize ( void )
{
  // PRNG
  sfmt_t rng;
  sfmt_init_gen_rand ( &rng, 23 );

  // Output file
  file_spike = fopen ( "spike.dat", "w" );

  // Cell parameters
  for ( int i = 0; i < N; i++ ) {
    v [ i ] = V_INIT + 10. * sfmt_genrand_real2 ( &rng );
    ge [ i ] = 0.;
    gi [ i ] = 0.;
    spike [ i ] = 0;
  }

  hipMalloc ( &d_v, N * sizeof ( double ) );
  hipMalloc ( &d_ge, N * sizeof ( double ) );
  hipMalloc ( &d_gi, N * sizeof ( double ) );
  hipMalloc ( &d_spike, N * sizeof ( int ) );

  hipMemcpy ( d_v, v, N * sizeof ( double ), hipMemcpyHostToDevice );
  hipMemcpy ( d_ge, ge, N * sizeof ( double ), hipMemcpyHostToDevice );
  hipMemcpy ( d_gi, gi, N * sizeof ( double ), hipMemcpyHostToDevice );
  hipMemcpy ( d_spike, spike, N * sizeof ( int ), hipMemcpyHostToDevice );

  // Synaptic weights
  w_exc = (int *) malloc ( N * N * sizeof ( int ) );
  w_inh = (int *) malloc ( N * N * sizeof ( int ) );

  for ( int i = 0; i < N; i++ ) {
    // From excitatory neurons to other neurons
    for ( int j = 0; j < N_EXC; j++ ) {
      w_exc [ j + N * i ] = ( sfmt_genrand_real2 ( &rng ) < P ) ? 1 : 0;
    }
    // From inhibitory neurons to other neurons
    for ( int j = N_EXC; j < N_EXC + N_INH; j++ ) {
      w_inh [ j + N * i ] = ( sfmt_genrand_real2 ( &rng ) < P ) ? 1 : 0;
    }
  }
  hipMalloc ( &d_w_exc, N * N * sizeof ( int ) );
  hipMalloc ( &d_w_inh, N * N * sizeof ( int ) );
  hipMemcpy ( d_w_exc, w_exc, N * N * sizeof ( int ), hipMemcpyHostToDevice );
  hipMemcpy ( d_w_inh, w_inh, N * N * sizeof ( int ), hipMemcpyHostToDevice );
}

void finalize ( void )
{
  fclose ( file_spike );
  free ( w_exc );
  free ( w_inh );
  hipFree ( d_v );
  hipFree ( d_ge );
  hipFree ( d_gi );
  hipFree ( d_spike );
  hipFree ( d_w_exc );
  hipFree ( d_w_inh );
}

__device__ void calculateSynapse ( int i, double *ge, double *gi, int *w_exc, int *w_inh, int *spike )
{
  if ( i < N )
  {
    double r = 0.;
    for ( int j = 0; j < N_EXC; j++ ){
      r += w_exc [ j + N * i ] * spike [ j ];
    }
    ge [ i ] += DT * ( G_EXC * r - ge [ i ] ) / TAU_GE;

    r = 0.;
    for ( int j = N_EXC; j < N_EXC + N_INH; j++ ){
      r += w_inh [ j + N * i ] * spike [ j ];
    }
    gi [ i ] += DT * ( G_INH * r - gi [ i ] ) / TAU_GI;
  }
}

__device__ void updateMembranePotential ( int i, double *v, double *ge, double *gi, int *spike )
{
  if ( i < N )
  {
    double dv = DT * ( - ( v [ i ] - V_LEAK ) + ge [ i ] + gi [ i ] ) / TAU_M;
    spike [ i ] = ( v [ i ] > THETA ) ? 1 : 0;
    v [ i ] = ( v [ i ] > THETA ) ? V_RESET : v [ i ] + dv;
  }
}

__global__ void kernel ( double *v, double *ge, double *gi, int *spike, int *w_exc, int *w_inh )
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;

  if ( i < N ) {
    calculateSynapse ( i, ge, gi, w_exc, w_inh, spike );
    updateMembranePotential ( i, v, ge, gi, spike );
  }
}

void outputSpike ( const double t )
{
  for ( int i = 0; i < N; i++ ) {
    if ( spike [ i ] ) { fprintf ( file_spike, "%f %d\n", t, i ); }
  }
}

void loop ( void )
{
  double t = 0.;
  timer_start ();

  int gridsize = ( N + BLOCK_SIZE - N % BLOCK_SIZE ) / BLOCK_SIZE;
  while ( t < T ) {
    kernel <<< gridsize, BLOCK_SIZE >>> ( d_v, d_ge, d_gi, d_spike, d_w_exc, d_w_inh );
    hipMemcpy ( spike, d_spike, N * sizeof ( int ), hipMemcpyDeviceToHost );
    outputSpike ( t );
    t = t + DT;
  }
  double elapsedTime = timer_elapsed ();
  printf ( "Elapsed time = %f sec.\n", elapsedTime);
}

int main ( void )
{
  initialize ();
  loop ();
  finalize ();

  return 0;
}
